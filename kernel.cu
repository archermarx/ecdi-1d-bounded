
#include <hip/hip_runtime.h>
template <typename T> __device__
inline T fwrap(const T x, const T len) {
    return x - len * floor(x / len);
}

template <typename T> __global__
void _wrap_particles(
    const int N,
    T* x, T* ux, T* uy, T* uz,
    const T* bulk_u, const T v_rms,
    const T* rand_vx, const T* rand_vy, const T* rand_vz,
    const T left_multiplier, const T right_multiplier,
    const T L_axial, const T dt
)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        x[i] = fma(ux[i], dt, x[i]);
        bool right = x[i] >= L_axial;
        bool left  = x[i] < 0;
        float sign = left ? -1.0 : 1.0;
        if (x[i] >= L_axial || x[i] < 0) {
            T v_rms_j = (left ? right_multiplier : left_multiplier) *  v_rms;
            x[i] = fwrap<T>(x[i], L_axial);
            ux[i] = fma(v_rms_j, rand_vx[i], bulk_u[0]);
            uy[i] = fma(v_rms_j, rand_vy[i], bulk_u[1]);
            uz[i] = fma(v_rms_j, rand_vz[i], bulk_u[2]);
        }
    }
}
